#include "hip/hip_runtime.h"
#include <vector>
#include <taskflow/taskflow.hpp>
#include <taskflow/cuda/cudaflow.hpp>

#include <overlap/gpu_overlap.hpp>

namespace gpu_overlap
{

    std::vector<int> flattenVector(const std::vector<std::vector<int>> &vec2D)
    {
        std::vector<int> vec1D;
        for (const auto &vec : vec2D)
        {
            vec1D.insert(vec1D.end(), vec.begin(), vec.end());
        }
        return vec1D;
    }

    std::vector<int> flattenVector(const std::vector<std::vector<std::vector<std::vector<int>>>> &vec4D)
    {
        std::vector<int> vec1D;
        for (const auto &vec3D : vec4D)
        {
            for (const auto &vec2D : vec3D)
            {
                for (const auto &vec : vec2D)
                {
                    vec1D.insert(vec1D.end(), vec.begin(), vec.end());
                }
            }
        }
        return vec1D;
    }

    std::vector<std::vector<int>> unflattenVector(const std::vector<int> &vec1D, size_t numRows, size_t numCols)
    {
        std::vector<std::vector<int>> vec2D(numRows, std::vector<int>(numCols));
        size_t index = 0;
        for (size_t i = 0; i < numRows; i++)
        {
            for (size_t j = 0; j < numCols; j++)
            {
                vec2D[i][j] = vec1D[index];
                index++;
            }
        }
        return vec2D;
    }

    std::vector<std::vector<std::vector<std::vector<int>>>> unflattenVector(const std::vector<int> &vec1D, size_t numLayers, size_t numChannels, size_t numRows, size_t numCols)
    {
        std::vector<std::vector<std::vector<std::vector<int>>>> vec4D(numLayers, std::vector<std::vector<std::vector<int>>>(numChannels, std::vector<std::vector<int>>(numRows, std::vector<int>(numCols))));
        size_t index = 0;
        for (size_t l = 0; l < numLayers; l++)
        {
            for (size_t c = 0; c < numChannels; c++)
            {
                for (size_t i = 0; i < numRows; i++)
                {
                    for (size_t j = 0; j < numCols; j++)
                    {
                        vec4D[l][c][i][j] = vec1D[index];
                        index++;
                    }
                }
            }
        }
        return vec4D;
    }

    ///-----------------------------------------------------------------------------
    ///
    /// sliding_window_kernel      A kernel function that performs a sliding window operation on a matrix.
    ///                            This kernel function oerates on a simualted 2D matrix, but the matrix is
    ///                            actually stored as a 1D array. The kernel function is designed to be
    ///                            launched with a 2D grid of 2D blocks. Each thread in the block will
    ///                            perform the sliding window operation on a single element in the input
    ///                            matrix. The output matrix will also be a 1D vector simulating a 4D vector with dimensions
    ///                            rows x cols x neigh_rows x neigh_cols.
    ///                            Each element at the output[i * cols + j] will be a 2D matrix (simulated by a flattened 1D vector)
    ///                            containing the neighbourhood of the input matrix element input[i * cols + j].
    ///
    /// @param[in] input           A pointer to the input matrix on the GPU.
    /// @param[out] output         A pointer to the output matrix on the GPU.
    /// @param[in] rows            The number of rows in the input matrix.
    /// @param[in] cols            The number of columns in the input matrix.
    /// @param[in] neib_rows       The number of rows in the neighbourhood.
    /// @param[in] neib_cols       The number of columns in the neighbourhood.
    /// @param[in] step_rows       The number of rows to step the neighbourhood over the input for each iteration.
    /// @param[in] step_cols       The number of columns to step the neighbourhood over the input for each iteration.
    /// @param[in] wrap_mode       A flag indicating whether the neighbourhood should wrap around the input matrix.
    /// @param[in] center_neigh    A flag indicating whether the neighbourhood should be centered over the current element in the input matrix.
    ///-----------------------------------------------------------------------------
    __global__ void sliding_window_kernel(int *input, int *output, int rows, int cols, int neib_rows, int neib_cols, int step_rows, int step_cols, bool wrap_mode, bool center_neigh)
    {
        // The thread index is the index of the element in the input matrix that the current thread will operate on.
        int i = blockIdx.y * blockDim.y + threadIdx.y; // Row index of the thread index
        int j = blockIdx.x * blockDim.x + threadIdx.x; // Column index of the thread index

        // The threads in the block that are outside the bounds of the input matrix do nothing.
        if (i < rows && j < cols)
        {
            // The output matrix is a 1D vector simulating a 4D vector with dimensions rows x cols x neigh_rows x neigh_cols.
            // ii and jj are the row and column indices of the current element in the neighbourhood.
            for (int ii = 0; ii < neib_rows; ++ii)
            {
                for (int jj = 0; jj < neib_cols; ++jj)
                {
                    // The indices of the current element in the neighbourhood.
                    int x = i + ii * step_rows; // Row index of the current element in the neighbourhood.
                    int y = j + jj * step_cols; // Column index of the current element in the neighbourhood.

                    // If the "center_neigh" flag is set, center the neighbourhood over the current element in the input matrix.
                    if (center_neigh)
                    {
                        x = i + (ii - neib_rows / 2) * step_rows;
                        y = j + (jj - neib_cols / 2) * step_cols;
                    }

                    // Wrap the indices around the bounds of the input matrix if "wrap_mode" is set.
                    if (wrap_mode)
                    {
                        x = (x + rows) % rows;
                        y = (y + cols) % cols;
                    }

                    // Set the element in the output matrix
                    if (x >= 0 && x < rows && y >= 0 && y < cols)
                    {
                        // Set output matrix element i,j,ii,jj to the input matrix element x,y.
                        int temp_idx = (i * cols + j) * neib_rows * neib_cols + ii * neib_cols + jj;
                        int temp_out = input[x * cols + y];
                        output[temp_idx] = temp_out;
                    }
                    else
                    {
                        // Set the element in the output matrix to 0 if the indices are outside the bounds of the input matrix.
                        int temp_idx = (i * cols + j) * neib_rows * neib_cols + ii * neib_cols + jj;
                        output[temp_idx] = 0;
                    }
                }
            }
        }
    }

    ///-----------------------------------------------------------------------------
    ///
    /// gpu_Images2Neibs           A function that performs a sliding window operation on a matrix.
    ///                            This function is designed to be called from the host. It allocates
    ///                            memory on the GPU, copies the input matrix to the GPU, launches the
    ///                            sliding_window_kernel kernel function, copies the output matrix from the GPU
    ///                            and frees the memory on the GPU.
    ///
    /// @param[in] input           A reference to the input matrix on the host. This is a 1D vector simulating a 2D matrix.
    /// @param[in] input_shape     A pair containing the number of rows and columns in the input matrix.
    /// @param[in] neib_shape      A pair containing the number of rows and columns in the neighbourhood.
    /// @param[in] neib_step       A pair containing the number of rows and columns to step the neighbourhood for each iteration.
    /// @param[in] wrap_mode       A flag indicating whether the neighbourhood should wrap around the input matrix.

    std::vector<int> gpu_Images2Neibs(
        const std::vector<int> &input,
        const std::pair<int, int> &input_shape,
        const std::pair<int, int> &neib_shape,
        const std::pair<int, int> &neib_step,
        bool wrap_mode,
        bool center_neigh)
    {
        // Determine the dimensions of the input matrix.
        const int rows = input_shape.first;
        const int cols = input_shape.second;

        // Check that the neighbourhood shape is valid.
        if (neib_shape.first > rows || neib_shape.second > cols)
        {
            throw std::invalid_argument("Neighbourhood shape must not be larger than the input matrix");
        }

        // Set the default step size to the neighbourhood shape.
        std::pair<int, int> step = neib_step;
        if (step.first == 0 && step.second == 0)
        {
            step = neib_shape;
        }

        int N = static_cast<int>(ceil(static_cast<float>(rows) / step.first));  // Number of rows in output matrix
        int M = static_cast<int>(ceil(static_cast<float>(cols) / step.second)); // Number of columns in output matrix
        int O = neib_shape.first;                                               // Number of rows in each patch
        int P = neib_shape.second;                                              // Number of columns in each patch

        // Create the output matrix. A 1D vector simulating a 4D vector with dimensions N x M x O x P.
        std::vector<int> output;

        // Allocate memory on the GPU for the input matrix.
        int *d_input, *d_output;

        tf::Taskflow taskflow("gpu_Images2Neibs");
        tf::Executor executor;

        // allocate device storage for the input matrix. The host (CPU) already has storage for the input.
        auto allocate_in = taskflow.emplace([&]()
                                            { TF_CHECK_CUDA(hipMalloc(&d_input, rows * cols * sizeof(int)), "failed to allocate input"); })
                               .name("allocate_in");

        // // allocate the host and device storage for the ouput matrix.
        auto allocate_out = taskflow.emplace([&]()
                                             {
                                                // Host storage
                                                output.resize(N * M * O * P);
                                                TF_CHECK_CUDA(hipMalloc(&d_output, N * M * O * P * sizeof(int)), "failed to allocate output"); })
                                .name("allocate_out");

        // create a cudaFlow to run the sliding_window_kernel.
        auto cudaFlow = taskflow.emplace([&](tf::cudaFlow &cf)
                                         {
                                            // copy the input matrix to the GPU. Copy from the first element in the multi dim vector.
                                            auto copy_in = cf.memcpy(d_input, input.data(), rows * cols * sizeof(int)).name("copy_in");

                                            // launch the kernel function on the GPU.
                                            int threadsPerBlock = 256;
                                            dim3 block(16, 16);   // 256 threads per block. A standard value this can be increased on some GPU models. 
                                            int noOfBlocks = cols * rows / 256;
                                            if ( (cols * rows) % threadsPerBlock) 
                                            {
                                                noOfBlocks++;
                                            }
                                            dim3 grid((cols + 16 - 1) / 16, (rows + 16 - 1) / 16);
                                            
                                            auto sliding_window = cf.kernel(grid, block, 0, sliding_window_kernel, d_input, d_output, rows, cols, neib_shape.first, neib_shape.second, step.first, step.second, wrap_mode, center_neigh)
                                                                        .name("sliding_window");

                                            // copy the output matrix back to the host. Copy to the pointer of the first element in the multi dim vector.
                                            auto copy_out = cf.memcpy(output.data(), d_output, N * M * O * P * sizeof(int) ).name("copy_out"); 
                                            sliding_window.succeed(copy_in)
                                                .precede(copy_out); })
                            .name("cudaFlow");

        auto free = taskflow.emplace([&]()
                                     {
                                         TF_CHECK_CUDA(hipFree(d_input), "failed to free d_input");
                                         TF_CHECK_CUDA(hipFree(d_output), "failed to free d_output"); })
                        .name("free");

        // create the dependency graph.
        cudaFlow.succeed(allocate_in, allocate_out)
            .precede(free);

        executor.run(taskflow)
            .wait();

        return output;
    }

} // namespace gpu_overlap